#include "hip/hip_runtime.h"
#include "Cuda.cuh"

__global__ void UpdateStateKernel(GrassState* currentStates, int* aliveCounts) {
    //int idx = blockIdx.x * blockDim.x + threadIdx.x;
    //if (idx < numCells) {
    //    UpdateState(currentStates[idx], aliveCounts[idx]);
    //}

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    GrassState currentState = currentStates[idx];
    int aliveCount = aliveCounts[idx];
    
    switch (currentState) {
    case startingStateAlive:
        if (!(aliveCount == 2 || aliveCount == 3)) {
            currentState = otherAliveCells;
        }
        else {
            currentState = aliveWith2or3Neighbors;
        }
        break;
    case startingStateDead:
        if (aliveCount < 3) {
            currentState = otherDeadCells;

        }
        else {
            currentState = deadWith3Neighbors;
        }
        break;
    default:
        break;
    }
    
    currentStates[idx] = currentState;    

}

hipError_t Cuda::CudaSetup()
{
    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//Error:
//
    return cudaStatus;

}

int minimum(int a, int b) {
    return (a < b) ? a : b;
}

hipError_t Cuda::UpdateStateCuda(GrassState* currentStates, int* aliveCounts, int sizeOfArray)
{
     // Allocates memory for currentStates and aliveCounts on the device
     GrassState* devCurrentStates;
     int* devAliveCounts;
    
     hipError_t cudaStatus;
    
     cudaStatus = hipMalloc((void**)&devCurrentStates, sizeOfArray * sizeof(GrassState));
     if (cudaStatus != hipSuccess) {
         fprintf(stderr, "hipMalloc failed!");
         return cudaStatus;
     }
    
     cudaStatus = hipMalloc((void**)&devAliveCounts, sizeOfArray * sizeof(int));
     if (cudaStatus != hipSuccess) {
         fprintf(stderr, "hipMalloc failed!");
         hipFree(devCurrentStates);
         return cudaStatus;
     }
    
     // Copy data from host to device
     cudaStatus = hipMemcpy(devCurrentStates, currentStates, sizeOfArray * sizeof(GrassState), hipMemcpyHostToDevice);
     if (cudaStatus != hipSuccess) {
         fprintf(stderr, "hipMemcpy failed!");
         hipFree(devCurrentStates);
         hipFree(devAliveCounts);
         return cudaStatus;
     }
    
     // Copy data from host to device for aliveCounts
     cudaStatus = hipMemcpy(devAliveCounts, aliveCounts, sizeOfArray * sizeof(int), hipMemcpyHostToDevice);
     if (cudaStatus != hipSuccess) {
         fprintf(stderr, "hipMemcpy failed!");
         hipFree(devCurrentStates);
         hipFree(devAliveCounts);
         return cudaStatus;
     }
    
     //int minimumGridSize;
     //int gridSize;
     //int blockSize;


     //hipOccupancyMaxPotentialBlockSize(&minimumGridSize, &blockSize, UpdateStateKernel, 0, sizeOfArray);
     //gridSize = (sizeOfArray + blockSize - 1) / blockSize;


     // Get device properties to determine optimal block size
     hipDeviceProp_t deviceProp;
     hipGetDeviceProperties(&deviceProp, 0); // Assuming device 0
     int maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;

     // Choose an optimal block size
     int sizeOfBlocks = minimum(maxThreadsPerBlock, 256); // Use the minimum function

     // Calculate the number of blocks
     int numOfBlocks = (sizeOfArray + sizeOfBlocks - 1) / sizeOfBlocks;
    
     ///Launch kernel
     UpdateStateKernel << <numOfBlocks, sizeOfBlocks >> > (devCurrentStates, devAliveCounts);

     // Check for any errors launching the kernel
     cudaStatus = hipGetLastError();
     if (cudaStatus != hipSuccess) {
         fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
         hipFree(devCurrentStates);
         hipFree(devAliveCounts);
         return cudaStatus;
     }
    
     // hipDeviceSynchronize waits for the kernel to finish
     // Returns any errors encountered during the launch.
     cudaStatus = hipDeviceSynchronize();
     if (cudaStatus != hipSuccess) {
         fprintf(stderr, "hipDeviceSynchronize failed: %s\n", hipGetErrorString(cudaStatus));
         hipFree(devCurrentStates);
         hipFree(devAliveCounts);
         return cudaStatus;
     }

       // Copy data from host to device
     cudaStatus = hipMemcpy(currentStates,devCurrentStates, sizeOfArray * sizeof(GrassState), hipMemcpyDeviceToHost);
     if (cudaStatus != hipSuccess) {
         fprintf(stderr, "hipMemcpy failed!");
         hipFree(devCurrentStates);
         hipFree(devAliveCounts);
         return cudaStatus;
     }
    
     hipFree(devCurrentStates);
     hipFree(devAliveCounts);
    
     return cudaStatus;
}